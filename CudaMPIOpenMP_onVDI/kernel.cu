#include "hip/hip_runtime.h"
#include "mainMethods.h"
#include "cudaKernel.h"
#define MAX_THREADS_PER_BLOCK 1000

//check with wich group point associated by f value
__device__ int getFSignCuda(double fVal)
{
	if (fVal >= 0)
		return GROUP1;
	else
		return GROUP2;
}

//one thread finds f for one point and the sign
__global__ void fOnGPUKernel(double *w, double *devicePoints, int *numOfDimensions, int *cudaPointsSigns)
{
	int i;
	int index = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	int fval = w[*numOfDimensions];

	for (i = 0; i < *numOfDimensions; i++)
	{
		fval += w[i] * devicePoints[index*(*numOfDimensions) + i];
	}
	cudaPointsSigns[index] = getFSignCuda(fval);
}

double* allocateWOnDevice(int numOfDimensions)
{
	double *deviceW = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "Failed to set Device inside allocateWOnDevice\n");

	cudaStatus = hipMalloc((void**)&deviceW, (numOfDimensions + 1) * sizeof(double));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "Failed to hipMalloc of w\n");

	return deviceW;
}

double *allocateAndCopyPointsToDevice(int numOfPoints, int numOfDimensions, double *points)
{
	hipError_t cudaStatus;
	double *devicePoints;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "Failed to set Device inside allocateAndCopyPointsToDevice\n");

	//start Allocate and copy points to CUDA
	cudaStatus = hipMalloc((void**)&devicePoints, numOfPoints*numOfDimensions * sizeof(double));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "Failed to hipMalloc of devicePoints\n");

	cudaStatus = hipMemcpy(devicePoints, points, numOfPoints*numOfDimensions * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "Failed to memCpy of points to device\n");
	//finish Allocate and copy points to CUDA
	return devicePoints;
}

int* allocateDevicePointsSigns(int numOfPoints)
{
	hipError_t cudaStatus;
	int *devicePointsSigns;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "Failed to set Device inside allocateDevicePointsSigns\n");

	//Allocate GPU buffer for device points signs
	cudaStatus = hipMalloc((void**)&devicePointsSigns, numOfPoints * sizeof(int));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "Failed to hipMalloc of devicePointsSigns\n");

	return devicePointsSigns;
}

int* allocateAndCopyNumOfDimensionsToDevice(int numOfDimensions)
{
	int *deviceNumOfDimensions;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "Failed to set Device inside allocateAndCopyNumOfDimensionsToDevice\n");

	//start Allocate and copy numOfDimensions to CUDA
	cudaStatus = hipMalloc((void**)&deviceNumOfDimensions, sizeof(int));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "Failed to hipMalloc of deviceNumOfDimensions\n");

	cudaStatus = hipMemcpy(deviceNumOfDimensions, &numOfDimensions, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "Failed to memCpy of numOfDimensions to device\n");
	//finish Allocate and copy numOfDimensions to CUDA

	return deviceNumOfDimensions;
}

//free cuda values at the end
void freeCudaValues(double *deviceW, double *devicePoints, int *devicePointsSigns, int *deviceNumOfDimensions)
{
	hipFree(deviceW);
	hipFree(devicePoints);
	hipFree(devicePointsSigns);
	hipFree(deviceNumOfDimensions);
}

hipError_t findPointGroupsforWByCuda(double *w, double *deviceW, double *devicePoints, int *devicePointsSigns, int *deviceNumOfDimensions,
	int numOfPoints, int numOfDimensions, int **pointsGroupsResult)
{
	int numOfBlocks = numOfPoints / MAX_THREADS_PER_BLOCK + 1;
	int *tempPointsGroupResult = 0;

	hipError_t cudaStatus = hipSuccess;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

	//start Copy weights to CUDA
	cudaStatus = hipMemcpy(deviceW, w, (numOfDimensions + 1) * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "CudaMemCpy of deviceW failed!");
	//finish Copy w to CUDA

	/***********************************************************************************/
	//perform f on gpu using number of blocks with 1000 threads
	fOnGPUKernel << <numOfBlocks, MAX_THREADS_PER_BLOCK >> > (deviceW, devicePoints, deviceNumOfDimensions, devicePointsSigns);
	//printf("\nfinished perform f on gpU using number of blocks with 1000 threads\n");
	/***********************************************************************************/

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "fOnGPUKernel launch failed");

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "Cuda Sync10launch failed");

	// Copy devicePointsSigns from GPU buffer to host memory.
	tempPointsGroupResult = (int*)malloc(numOfPoints * sizeof(int));
	cudaStatus = hipMemcpy((void *)tempPointsGroupResult, (void *)(devicePointsSigns), numOfPoints * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "CudaMemCpy of tempPointsGroupResult failed");
	(*pointsGroupsResult) = tempPointsGroupResult;
	// End Copy devicePointsSigns from GPU buffer to host memory.

	return cudaStatus;
}